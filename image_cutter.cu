
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>


_global__ void imageCutter (int* fotoOriginal, int* fotoNueva, int posFotoNueva, int fotoSize, int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < fotoSize*fotoSize){
    	fotoNueva[tid] = fotoOriginal[posFotoNueva+tid%fotoSize+(tid/fotoSize)*N];
    }
}


// si queremos calcular el algoritmo en todos los puntos posibles (una combinacion por kernel), tendremos que correr 4*fotoSize² kernels distintos

// Para que el algoritmo sea mas optimo, se pueden normalizar las imagenes primero, esto permite que donde se encuentren puntos negros sea igual a encontrar puntos blancos




/* Ideas para optimizar:
- no probar todos los puntos, saltarse cada 4 por ejemplo y luego calcular los puntos cercanos al ganador 
- que cada kernel se encargue de varias combinaciones
	- se podria usar memoria comparitda ya que los datos deberian estar bien cercanos !!
	- Guardamos en memoria compartida la informacion que necesitemos para las N iteraciones
	- Generamos un espacio en memoria compartida para hacer la reduccion
	- Cada vez que realizemos una iteracion, hacemos la reduccion y guardamos el valor en memoria global
- downsample the two images first, that way processing time is lowered


*/


// acercamiento mas simple (y menos optimizado)
_global__ void crossCorrelation (int* foto1, int* foto2, int pixelCount, int Iteracion, int minX, int maxX, int minY, int maxY){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    




}








